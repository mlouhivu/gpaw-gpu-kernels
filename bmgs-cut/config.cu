#include "hip/hip_runtime.h"
#include "config.h"

t_arg as_arg(int layers, int3 dimx, int3 dimy, int3 position)
{
    t_arg arg;

    arg.layers = layers;
    arg.dimx = dimx;
    arg.dimy = dimy;
    arg.position = position;

    return arg;
}

t_config get_config()
{
    t_config config;
    config.nargs = 0;

    int layers;
    int3 dimx;
    int3 dimy;
    int3 position;

    // carbon nanotube
    layers = 56;
    dimx = {41,21,32};
    dimy = {41,21,1};
    position = {0,0,0};
    config.args[config.nargs++] = as_arg(layers, dimx, dimy, position);

    layers = 56;
    dimx = {85,46,68};
    dimy = {79,40,3};
    position = {3,3,62};
    config.args[config.nargs++] = as_arg(layers, dimx, dimy, position);

    layers = 56;
    dimx = {85,45,68};
    dimy = {79,39,3};
    position = {3,3,3};
    config.args[config.nargs++] = as_arg(layers, dimx, dimy, position);

    layers = 56;
    dimx = {21,11,17};
    dimy = {19,1,15};
    position = {1,9,1};
    config.args[config.nargs++] = as_arg(layers, dimx, dimy, position);

    layers = 56;
    dimx = {21,11,18};
    dimy = {19,9,1};
    position = {1,1,1};
    config.args[config.nargs++] = as_arg(layers, dimx, dimy, position);

    // copper filament
    layers = 25;
    dimx = {89,52,62};
    dimy = {83,46,3};
    position = {3,3,56};
    config.args[config.nargs++] = as_arg(layers, dimx, dimy, position);

    layers = 25;
    dimx = {43,24,29};
    dimy = {43,24,1};
    position = {0,0,0};
    config.args[config.nargs++] = as_arg(layers, dimx, dimy, position);

    layers = 25;
    dimx = {43,25,30};
    dimy = {41,24,28};
    position = {1,1,1};
    config.args[config.nargs++] = as_arg(layers, dimx, dimy, position);

    layers = 48;
    dimx = {89,52,62};
    dimy = {83,46,3};
    position = {3,3,56};
    config.args[config.nargs++] = as_arg(layers, dimx, dimy, position);

    // single fullerene
    layers = 1;
    dimx = {6,7,12};
    dimy = {1,5,11};
    position = {0,1,0};
    config.args[config.nargs++] = as_arg(layers, dimx, dimy, position);

    layers = 1;
    dimx = {12,12,23};
    dimy = {1,11,22};
    position = {0,0,0};
    config.args[config.nargs++] = as_arg(layers, dimx, dimy, position);

    // other
    layers = 12;
    dimx = {252,31,64};
    dimy = {252,31,1};
    position = {0,0,0};
    config.args[config.nargs++] = as_arg(layers, dimx, dimy, position);

    layers = 8;
    dimx = {100,100,100};
    dimy = {10,10,10};
    position = {22,44,66};
    config.args[config.nargs++] = as_arg(layers, dimx, dimy, position);

    return config;
}

void get_kernels(kernel_func *kernels)
{
    for (int j=0; j < MAX_KERNELS; j++)
        kernels[j] = NULL;

    int i = 0;
    kernels[i++] = &run_kernel0;
    kernels[i++] = &run_kernel1;
    kernels[i++] = &run_kernel2;
    kernels[i++] = &run_kernel2b;
    kernels[i++] = &run_kernel3;
    kernels[i++] = &run_kernel3b;
    kernels[i++] = &run_kernel3c;
    kernels[i++] = &run_kernel4;
    kernels[i++] = &run_kernel4b;
    kernels[i++] = &run_kernel5;
}
