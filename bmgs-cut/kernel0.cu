#include "hip/hip_runtime.h"
#include "kernels.h"


__global__ void Zcuda(bmgs_cut_cuda_kernel)(
        const Tcuda* a, const int3 c_sizea, Tcuda* b, const int3 c_sizeb,
        int blocks, int xdiv)
{
    int xx = gridDim.x / xdiv;  // xdiv == x ; xx == Nz
    int yy = gridDim.y / blocks; // blocks == blocks ; yy == Ny

    int blocksi = blockIdx.y / yy;  // blockID for blocks
    int i1 = (blockIdx.y - blocksi * yy) * blockDim.y + threadIdx.y;
      // gid in y from block start

    int xind = blockIdx.x / xx;  // blockID for x
    int i2 = (blockIdx.x - xind * xx) * blockDim.x + threadIdx.x;
      // gid in z from x start

    b += i2 + (i1 + (xind + blocksi * c_sizeb.x) * c_sizeb.y) * c_sizeb.z;
    a += i2 + (i1 + (xind + blocksi * c_sizea.x) * c_sizea.y) * c_sizea.z;

/*    b += i2
       + i1      * c_sizeb.z
       + xind    * c_sizeb.y * c_sizeb.z;
       + blocksi * c_sizeb.x * c_sizeb.y * c_sizeb.z;
*/
    while (xind < c_sizeb.x) {
        if ((i2 < c_sizeb.z) && (i1 < c_sizeb.y)) {
            b[0] = a[0];
        }
        b += xdiv * c_sizeb.y * c_sizeb.z;
        a += xdiv * c_sizea.y * c_sizea.z;
        xind += xdiv;
    }
}


void Zcuda(bmgs_cut_cuda_gpu)(
        const Tcuda* a, const int sizea[3], const int starta[3],
        Tcuda* b, const int sizeb[3],
        int blocks, dim3 *blx, dim3 *thx)
{
    if (!(sizea[0] && sizea[1] && sizea[2]))
        return;

    int3 hc_sizea, hc_sizeb;
    hc_sizea.x=sizea[0];
    hc_sizea.y=sizea[1];
    hc_sizea.z=sizea[2];
    hc_sizeb.x=sizeb[0];
    hc_sizeb.y=sizeb[1];
    hc_sizeb.z=sizeb[2];

#ifdef DEBUG_CUDA_CUT
    int ng = sizea[0] * sizea[1] * sizea[2];
    int ng2 = sizeb[0] * sizeb[1] * sizeb[2];
    double* a_cpu = GPAW_MALLOC(double, ng * blocks);
    double* b_cpu = GPAW_MALLOC(double, ng2 * blocks);
    double* a_cpu2 = GPAW_MALLOC(double, ng * blocks);
    double* b_cpu2 = GPAW_MALLOC(double, ng2 * blocks);
    const Tcuda* a2 = a;

    GPAW_CUDAMEMCPY(a_cpu, a, double, ng * blocks, hipMemcpyDeviceToHost);
    GPAW_CUDAMEMCPY(b_cpu, b, double, ng2 * blocks, hipMemcpyDeviceToHost);
#endif //DEBUG_CUDA_CUT

    int blockx = MIN(nextPow2(hc_sizeb.z), BLOCK_MAX);
    int blocky = MIN(
            MIN(nextPow2(hc_sizeb.y), BLOCK_TOTALMAX / blockx),
            BLOCK_MAX);
    dim3 dimBlock(blockx, blocky);
    int gridx = ((hc_sizeb.z + dimBlock.x - 1) / dimBlock.x);
    int xdiv = MAX(1, MIN(hc_sizeb.x, GRID_MAX / gridx));
    int gridy = blocks * ((hc_sizeb.y + dimBlock.y - 1) / dimBlock.y);

    gridx = xdiv * gridx;
    dim3 dimGrid(gridx, gridy);

    thx->x = blockx;
    thx->y = blocky;
    blx->x = gridx;
    blx->y = gridy;

    a += starta[2] + (starta[1] + starta[0] * hc_sizea.y) * hc_sizea.z;

    Zcuda(bmgs_cut_cuda_kernel)<<<dimGrid, dimBlock, 0>>>(
            (Tcuda*) a, hc_sizea, (Tcuda*) b, hc_sizeb,
         blocks, xdiv);
    //gpaw_cudaSafeCall(hipGetLastError());

#ifdef DEBUG_CUDA_CUT
    for (int m=0; m < blocks; m++) {
        bmgs_cut(a_cpu + m * ng, sizea, starta, b_cpu + m * ng2, sizeb);
    }
    hipDeviceSynchronize();
    GPAW_CUDAMEMCPY(a_cpu2, a2, double, ng * blocks,
            hipMemcpyDeviceToHost);
    GPAW_CUDAMEMCPY(b_cpu2, b, double, ng2 * blocks,
            hipMemcpyDeviceToHost);

    double a_err = 0;
    double b_err = 0;
    for (int i=0; i < ng2 * blocks; i++) {
        b_err = MAX(b_err, fabs(b_cpu[i] - b_cpu2[i]));
        if (i < ng * blocks) {
            a_err = MAX(a_err, fabs(a_cpu[i] - a_cpu2[i]));
        }
    }
    if ((b_err > GPAW_CUDA_ABS_TOL_EXCT)
            || (a_err > GPAW_CUDA_ABS_TOL_EXCT)) {
        fprintf(stderr, "Debug cuda cut errors: a %g b %g\n",
                a_err, b_err);
    }
    free(a_cpu);
    free(b_cpu);
    free(a_cpu2);
    free(b_cpu2);
#endif //DEBUG_CUDA_CUT
}


float run_kernel0(double *x_, const int3 sizex, const int3 pos,
                  double *y_, const int3 sizey,
                  const unsigned int layers,
                  dim3 blx, dim3 thx, char *title, char *header)
{
    const int dimx[3] = {sizex.x, sizex.y, sizex.z};
    const int dimy[3] = {sizey.x, sizey.y, sizey.z};
    const int position[3] = {pos.x, pos.y, pos.z};

    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    double *xx_;
    double *yy_;

    char name[32];

    xx_ = x_;
    yy_ = y_;
    hipEventRecord(start);
    bmgs_cut_cuda_gpu(xx_, dimx, position, yy_, dimy, layers, &blx, &thx);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    sprintf(name, "KERNEL");
    sprintf(title, "%s %8s", title, name);
    sprintf(header, "%s  <<<(%d,%d,%d), (%d, %d, %d)>>>", name,
            blx.x, blx.y, blx.z, thx.x, thx.y, thx.z);
    return time;
}
