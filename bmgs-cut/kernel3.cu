#include "hip/hip_runtime.h"
#include "kernels.h"

__global__ void Zcuda(bmgs_cut_cuda_kernel3)(
        Tcuda *src, Tcuda *tgt, int3 n, int3 m, int3 o, const Tcuda phase)
{
    int tidx = threadIdx.x + blockIdx.x * blockDim.x;
    int tidy = threadIdx.y + blockIdx.y * blockDim.y;
    int tidz = threadIdx.z;
    int stridex = gridDim.x * blockDim.x;
    int stridey = gridDim.y * blockDim.y;
    int stridez = blockDim.z;
    int b = blockIdx.z;
    int t, s, tz, sz, tb, sb;
    int i, j, k;

    tb = m.z * m.y * m.x * b;
    sb = n.z * n.y * n.x * b;
    for (i = tidz; i < m.x; i += stridez) {
        tz = tb + m.z * m.y * i;
        sz = sb + n.z * n.y * (i + o.x) + o.z;
        for (j = tidy; j < m.y; j += stridey) {
            t = tz + m.z * j;
            s = sz + n.z * (j + o.y);
            for (k = tidx; k < m.z; k += stridex) {
                tgt[k + t] = MULTT(phase, src[k + s]);
            }
        }
    }
}

/*** New GPU implementation (multi-block, block in dim) ***/
float run_kernel3(Tcuda *x_, const int3 sizex, const int3 pos,
                  Tcuda *y_, const int3 sizey, const int layers,
                  const Tcuda phase_,
                  char *title, char *header,
                  const int repeat, const int trial)
{
    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    dim3 blocks, threads;

    char name[32];

    hipEventRecord(start);
    for (int i=0; i < repeat; i++) {
        threads.x = MIN(nextPow2(sizey.z), BLOCK_TOTALMAX);
        threads.y = MIN(nextPow2(sizey.y), BLOCK_TOTALMAX / threads.x);
        threads.z = BLOCK_TOTALMAX / (threads.x * threads.y);
        blocks.x = (sizey.z + threads.x - 1) / threads.x;
        blocks.y = (sizey.y + threads.y - 1) / threads.y;
        blocks.z = layers;
        Zcuda(bmgs_cut_cuda_kernel3)<<<blocks, threads>>>(
                x_, y_, sizex, sizey, pos, phase_);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    sprintf(name, "KERNEL3");
    if (!trial)
        sprintf(title, "%s %8s", title, name);
    sprintf(header, "%s  <<<(%d,%d,%d), (%d, %d, %d)>>>", name,
            blocks.x, blocks.y, blocks.z, threads.x, threads.y, threads.z);
    return time;
}

float run_kernel3b(Tcuda *x_, const int3 sizex, const int3 pos,
                   Tcuda *y_, const int3 sizey, const int layers,
                   const Tcuda phase_,
                   char *title, char *header,
                   const int repeat, const int trial)
{
    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    dim3 blocks, threads;

    char name[32];

    hipEventRecord(start);
    for (int i=0; i < repeat; i++) {
        threads.x = MIN(nextPow2(sizey.z), BLOCK_MAX);
        threads.y = MIN(nextPow2(sizey.y), BLOCK_TOTALMAX / threads.x);
        threads.z = BLOCK_TOTALMAX / (threads.x * threads.y);
        blocks.x = (sizey.z + threads.x - 1) / threads.x;
        blocks.y = (sizey.y + threads.y - 1) / threads.y;
        blocks.z = layers;
        Zcuda(bmgs_cut_cuda_kernel3)<<<blocks, threads>>>(
                x_, y_, sizex, sizey, pos, phase_);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    sprintf(name, "KERN3v2");
    if (!trial)
        sprintf(title, "%s %8s", title, name);
    sprintf(header, "%s  <<<(%d,%d,%d), (%d, %d, %d)>>>", name,
            blocks.x, blocks.y, blocks.z, threads.x, threads.y, threads.z);
    return time;
}

float run_kernel3c(Tcuda *x_, const int3 sizex, const int3 pos,
                   Tcuda *y_, const int3 sizey, const int layers,
                   const Tcuda phase_,
                   char *title, char *header,
                   const int repeat, const int trial)
{
    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    dim3 blocks, threads;

    char name[32];

    hipEventRecord(start);
    for (int i=0; i < repeat; i++) {
        threads.x = MIN(nextPow2(sizey.z), BLOCK_MAX);
        threads.y = MIN(nextPow2(sizey.y), BLOCK_MAX / threads.x);
        threads.z = BLOCK_MAX / (threads.x * threads.y);
        blocks.x = (sizey.z + threads.x - 1) / threads.x;
        blocks.y = (sizey.y + threads.y - 1) / threads.y;
        blocks.z = layers;
        Zcuda(bmgs_cut_cuda_kernel3)<<<blocks, threads>>>(
                x_, y_, sizex, sizey, pos, phase_);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    sprintf(name, "KERN3v3");
    if (!trial)
        sprintf(title, "%s %8s", title, name);
    sprintf(header, "%s  <<<(%d,%d,%d), (%d, %d, %d)>>>", name,
            blocks.x, blocks.y, blocks.z, threads.x, threads.y, threads.z);
    return time;
}
