#include "hip/hip_runtime.h"
#include "kernels.h"

__global__ void Zcuda(bmgs_cut_cuda_kernel4)(
        Tcuda *src, Tcuda *tgt, int3 n, int3 m, int3 o, int blocks)
{
    int gridsize_y = (gridDim.y + blocks - 1) / blocks;
    int b = blockIdx.y / gridsize_y;
    int tidx = threadIdx.x + blockIdx.x * blockDim.x;
    int tidy = threadIdx.y + (blockIdx.y - b * gridsize_y) * blockDim.y;
    int tidz = threadIdx.z;
    int stridex = gridDim.x * blockDim.x;
    int stridey = gridsize_y * blockDim.y;
    int stridez = blockDim.z;
    int t, s, tz, sz, tb, sb;
    int i, j, k;

    tb = m.z * m.y * m.x * b;
    sb = n.z * n.y * n.x * b;
    for (i = tidz; i < m.x; i += stridez) {
        tz = tb + m.z * m.y * i;
        sz = sb + n.z * n.y * (i + o.x) + o.z;
        for (j = tidy; j < m.y; j += stridey) {
            t = tz + m.z * j;
            s = sz + n.z * (j + o.y);
            for (k = tidx; k < m.z; k += stridex) {
                tgt[k + t] = src[k + s];
            }
        }
    }
}

/*** New GPU implementation (multi-block, block in dim) ***/
float run_kernel4(double *x_, const int3 sizex, const int3 pos,
                  double *y_, const int3 sizey,
                  const unsigned int layers,
                  char *title, char *header, const int repeat)
{
    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    dim3 blocks, threads;

    char name[32];

    hipEventRecord(start);
    threads.x = MIN(nextPow2(sizey.z), BLOCK_MAX);
    threads.y = MIN(MIN(nextPow2(sizey.y), BLOCK_TOTALMAX / threads.x),
                    BLOCK_MAX);
    threads.z = MIN(BLOCK_TOTALMAX / (threads.x * threads.y), BLOCK_MAX);
    blocks.x = (sizey.z + threads.x - 1) / threads.x;
    blocks.y = layers * ((sizey.y + threads.y - 1) / threads.y);
    blocks.z = 1;
    bmgs_cut_cuda_kernel4<<<blocks, threads>>>(
            x_, y_, sizex, sizey, pos, layers);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    sprintf(name, "KERNEL4");
    if (!repeat)
        sprintf(title, "%s %8s", title, name);
    sprintf(header, "%s  <<<(%d,%d,%d), (%d, %d, %d)>>>", name,
            blocks.x, blocks.y, blocks.z, threads.x, threads.y, threads.z);
    return time;
}

float run_kernel4b(double *x_, const int3 sizex, const int3 pos,
                   double *y_, const int3 sizey,
                   const unsigned int layers,
                   char *title, char *header, const int repeat)
{
    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    dim3 blocks, threads;

    char name[32];

    hipEventRecord(start);
    threads.x = 1;
    threads.y = MIN(nextPow2(sizey.y), BLOCK_MAX);
    threads.z = MIN(BLOCK_MAX / (threads.x * threads.y), BLOCK_MAX);
    blocks.x = (sizey.z + threads.x - 1) / threads.x;
    blocks.y = layers * ((sizey.y + threads.y - 1) / threads.y);
    blocks.z = 1;
    bmgs_cut_cuda_kernel4<<<blocks, threads>>>(
            x_, y_, sizex, sizey, pos, layers);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    sprintf(name, "KERN4v2");
    if (!repeat)
        sprintf(title, "%s %8s", title, name);
    sprintf(header, "%s  <<<(%d,%d,%d), (%d, %d, %d)>>>", name,
            blocks.x, blocks.y, blocks.z, threads.x, threads.y, threads.z);
    return time;
}
